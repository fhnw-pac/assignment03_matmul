﻿#include <iostream>
#include <stdlib.h>
#include "hip/hip_runtime.h"

#include <omp.h>
#include <chrono>

using namespace std;


/* This is our CUDA call wrapper, we will use in PAC.
*
*  Almost all CUDA calls should be wrapped with this makro.
*  Errors from these calls will be catched and printed on the console.
*  If an error appears, the program will terminate.
*
* Example: gpuErrCheck(cudaMalloc(&deviceA, N * sizeof(int)));
*          gpuErrCheck(cudaMemcpy(deviceA, hostA, N * sizeof(int), cudaMemcpyHostToDevice));
*/
#define gpuErrCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        std::cout << "GPUassert: " << hipGetErrorString(code) << " " << file << " " << line << std::endl;
        if (abort)
        {
            exit(code);
        }
    }
}


/* Helper which populates a matrix buffer (dimSize*dimSize).
* 
* Think of this as it would load the data from disk or somewhere else.
* This dummy data is only used to fill the buffer as fast as possible.
*/
void populateMatrixBuffer(float* buffer, int dimSize)
{
    // Init of matrix buffer
    for (int i = 0; i < dimSize; i++) {
        for (int j = 0; j < dimSize; j++) {
            buffer[i * dimSize + j] = 1.0f / j;
        }
    }
}


// Compare result arrays CPU vs GPU result. If no diff, the result pass.
int compareResultVec(float* matrixCPU, float* matrixGPU, int size)
{
    float error = 0;
    for (int i = 0; i < size; i++)
    {
        error += abs(matrixCPU[i] - matrixGPU[i]);
    }
    if (error == 0)  // Is this sane? Think about float processing!
    {
        cout << "Test passed." << endl;
        return 0;
    }
    else
    {
        cout << "Accumulated error: " << error << endl;
        return -1;
    }
}


/* Slow MatMul on the CPU, stores matrixA * matrixB in buffer matrixC
* 
* This is our CPU baseline.
*/
void matMulCPUNaive(float* matrixA, float* matrixB, float* matrixC, int dimSize)
{
    float sum;
    for (int i = 0; i < dimSize; i++)
    {
        for (int j = 0; j < dimSize; j++)
        {
            sum = 0.0;
            for (int n = 0; n < dimSize; n++)
            {
                sum += matrixA[i * dimSize + n] * matrixB[n * dimSize + j];
            }
            matrixC[i * dimSize + j] = sum;
        }
    }
}


int main()
{
    // ATTENTION: Your code must be robust in regards of this number.
    // ATTENTION: DIM_SIZE of 4096 is maybe not a good idea during development :)
    // DIM_SIZE can and will change during the assessment, also to non 2^n values!
    for (int DIM_SIZE = 64; DIM_SIZE <= 4096; DIM_SIZE <<= 1) {
        cout << "DIM_SIZE: " << DIM_SIZE << endl;
        float* h_matrixA = new float[DIM_SIZE * DIM_SIZE];
        float* h_matrixB = new float[DIM_SIZE * DIM_SIZE];
        float* h_matrixC = new float[DIM_SIZE * DIM_SIZE];
        populateMatrixBuffer(h_matrixA, DIM_SIZE);
        populateMatrixBuffer(h_matrixB, DIM_SIZE);

        auto startTime = chrono::high_resolution_clock::now();
        matMulCPUNaive(h_matrixA, h_matrixB, h_matrixC, DIM_SIZE);
        auto endTime = chrono::high_resolution_clock::now();
        cout << "CPU time [ms]: " << chrono::duration_cast<chrono::milliseconds>(endTime - startTime).count() << endl;

        delete[] h_matrixA;
        delete[] h_matrixB;
        delete[] h_matrixC;
    }

    return 0;
}
